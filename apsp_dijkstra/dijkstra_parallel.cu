#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <limits.h>
#include <hip/hip_runtime.h>


#define TRUE 1
#define FALSE 0
#define INFNTY INT_MAX

typedef int boolean;

/* Generates a random undirected graph represented by an adjacency matrix */
void generate_random_graph(int V, int *adjacency_matrix)
{
    srand(time(NULL));

    for (int i = 0; i < V; i++)
    {
        for (int j = 0; j < V; j++)
        {
            if (i != j)
            {
                adjacency_matrix[i * V + j] = rand() % 10;                 /* Assign a random value corresponding to the edge */
                adjacency_matrix[j * V + i] = adjacency_matrix[i * V + j]; /* Graph is undirected, the adjacency matrix is symmetric */
            }
            else
            {
                adjacency_matrix[i * V + j] = 0;
            }
        }
    }
}

__global__ void dijkstra_kernel(int V, int *graph, int *len, int *temp_distance, boolean *visited)
{
    int source = blockIdx.x * blockDim.x + threadIdx.x;

    if (source < V)
    {
        for (int i = 0; i < V; ++i)
        {
            visited[i] = FALSE;
            temp_distance[i] = INFNTY;
            len[source * V + i] = INFNTY;
        }

        len[source * V + source] = 0;

        for (int count = 0; count < V - 1; ++count)
        {
            int current_vertex = -1;
            int min_distance = INFNTY;

            for (int v = 0; v < V; ++v)
            {
                if (!visited[v] && len[source * V + v] <= min_distance)
                {
                    min_distance = len[source * V + v];
                    current_vertex = v;
                }
            }

            visited[current_vertex] = TRUE;

            for (int v = 0; v < V; ++v)
            {
                int weight = graph[current_vertex * V + v];
                if (!visited[v] && weight && len[source * V + current_vertex] != INFNTY &&
                    len[source * V + current_vertex] + weight < len[source * V + v])
                {
                    len[source * V + v] = len[source * V + current_vertex] + weight;
                    temp_distance[v] = len[source * V + v];
                }
            }
        }
    }
}

void dijkstra_parallel(int V, int *adjacency_matrix, int *len, int *temp_distance)
{
    boolean *d_visited;
    int *d_len, *d_temp_distance, *d_adjacency_matrix;

    /* Allocate memory on GPU */
    hipMalloc((void **)&d_visited, V * sizeof(boolean));
    hipMalloc((void **)&d_len, V * V * sizeof(int));
    hipMalloc((void **)&d_temp_distance, V * sizeof(int));
    hipMalloc((void **)&d_adjacency_matrix, V * V * sizeof(int));

    /* Copy data to GPU */
    hipMemcpy(d_adjacency_matrix, adjacency_matrix, V * V * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 gridSize((V + blockSize.x - 1) / blockSize.x);

    clock_t start = clock(); /* Start timer */

    /* Launch CUDA kernel */
    dijkstra_kernel<<<gridSize, blockSize>>>(V, d_adjacency_matrix, d_len, d_temp_distance, d_visited);

    hipDeviceSynchronize(); /* Sync GPU and CPU to ensure kernel finished */

    /* Copy results back to CPU */
    hipMemcpy(len, d_len, V * V * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(temp_distance, d_temp_distance, V * sizeof(int), hipMemcpyDeviceToHost);

    clock_t end = clock();
    float seconds = (float)(end - start) / CLOCKS_PER_SEC;
    printf("TOTAL ELAPSED TIME ON GPU = %f SECS\n", seconds);

    /* Free allocated memory on GPU */
    hipFree(d_visited);
    hipFree(d_len);
    hipFree(d_temp_distance);
    hipFree(d_adjacency_matrix);
}

int main(int argc, char **argv)
{
    if (argc != 2)
    {
        printf("USAGE: ./dijkstra_parallel <number_of_vertices>\n");
        return 1;
    }

    int *len, *temp_distance;
    int V = atoi(argv[1]); /* Number of vertices */

    len = (int *)malloc(V * V * sizeof(int));
    temp_distance = (int *)malloc(V * sizeof(int));

    int *adjacency_matrix = (int *)malloc(V * V * sizeof(int));

    generate_random_graph(V, adjacency_matrix);
    dijkstra_parallel(V, adjacency_matrix, len, temp_distance);

    free(len);
    free(temp_distance);
    free(adjacency_matrix);

    return 0;
}
