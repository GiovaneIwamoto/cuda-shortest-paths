#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <limits.h>
#include <hip/hip_runtime.h>


#define INFNTY INT_MAX

int **adjacency_matrix, **dp_matrix;
int n_vertices;

/* Undirected graph non-negative edge weights */
void generate_random_adj_matrix(int n_vertices)
{
    int N = n_vertices;
    int i, j;

    /* Allocate memory for adjacency matrix 2D array */
    adjacency_matrix = (int **)malloc(N * sizeof(int *));

    for (i = 0; i < N; i++)
    {
        adjacency_matrix[i] = (int *)malloc(N * sizeof(int));
    }
    srand(0);

    for (i = 0; i < N; i++)
    {
        for (j = i; j < N; j++)
        {
            if (i == j)
            {
                adjacency_matrix[i][j] = 0;
            }
            else
            {
                /* Zero to nine random */
                int r = rand() % 10;
                int val = (r == 2) ? INFNTY : r; /* No edge between vertices */
                adjacency_matrix[i][j] = val;    /* Symmetrically */
                adjacency_matrix[j][i] = val;    /* Symmetrically */
            }
        }
    }
}

void floyd_warshall_serial(int **graph, int **dp, int N)
{
    int i, j, k;
    /* Initialize copy graph to dp matrix */
    for (i = 0; i < N; i++)
        for (j = 0; j < N; j++)
            dp[i][j] = graph[i][j];

    clock_t start = clock(); /* Start measuring execution time */

    /* Floyd Warshall algorithm */
    for (k = 0; k < N; k++)
    {
        for (i = 0; i < N; i++)
        {
            for (j = 0; j < N; j++)
            {
                if (dp[i][k] + dp[k][j] < dp[i][j])
                    dp[i][j] = dp[i][k] + dp[k][j];
            }
        }
    }

    /* Stop measuring execution time */
    clock_t end = clock();
    float cpu_time = (float)(end - start) / CLOCKS_PER_SEC;

    /* Save serial results to log file */
    FILE *file = fopen("floyd_results.log", "a");
    fprintf(file, "Floyd CPU | Execution Time: %f seconds | Size: %d\n", cpu_time, n_vertices);

    fclose(file);
}

/* Kernel CUDA Floyd algorithm */
__global__ void floyd_warshall_kernel(int *dev_dp, int n_vertices, int k)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n_vertices && j < n_vertices)
    {
        int ij_index = i * n_vertices + j;
        int ik_index = i * n_vertices + k;
        int kj_index = k * n_vertices + j;

        if (dev_dp[ik_index] != INFNTY && dev_dp[kj_index] != INFNTY)
        {
            int sum = dev_dp[ik_index] + dev_dp[kj_index];
            if (sum < dev_dp[ij_index])
            {
                dev_dp[ij_index] = sum;
            }
        }
    }
}

// Function to initialize GPU and run Floyd-Warshall algorithm
void floyd_warshall_parallel(int **adj_matrix, int **dp_matrix, int n_vertices)
{
    dim3 blockSize(16, 16);
    dim3 gridSize((n_vertices + blockSize.x - 1) / blockSize.x, (n_vertices + blockSize.y - 1) / blockSize.y);

    int *dev_dp;
    hipMalloc((void **)&dev_dp, n_vertices * n_vertices * sizeof(int));

    for (int i = 0; i < n_vertices; i++)
    {
        hipMemcpy(dev_dp + i * n_vertices, adj_matrix[i], n_vertices * sizeof(int), hipMemcpyHostToDevice);
    }

    clock_t start = clock(); /* Start measuring execution time */

    /* Execute kernel for each vertex, k pivot */
    for (int k = 0; k < n_vertices; k++)
    {
        floyd_warshall_kernel<<<gridSize, blockSize>>>(dev_dp, n_vertices, k);
        hipDeviceSynchronize(); /* Sync all kernels finished */
    }

    /* Stop measuring execution time */
    clock_t end = clock();
    float gpu_time = (float)(end - start) / CLOCKS_PER_SEC;

    /* Save parallel results to log file */
    FILE *file = fopen("floyd_results.log", "a");
    fprintf(file, "Floyd GPU | Execution Time: %f seconds | Size: %d\n", gpu_time, n_vertices);
    fclose(file);

    /* Return dp matrix to CPU */
    for (int i = 0; i < n_vertices; i++)
    {
        hipMemcpy(dp_matrix[i], dev_dp + i * n_vertices, n_vertices * sizeof(int), hipMemcpyDeviceToHost);
    }

    /* Free allocated memory */
    hipFree(dev_dp);
}

int main(int argc, char **argv)
{
    int i;
    if (argc != 2)
    {
        printf("USAGE: ./floyd_comparison <number_of_vertices>\n");
        return 1;
    }

    n_vertices = atoi(argv[1]);

    dp_matrix = (int **)malloc(n_vertices * sizeof(int *));
    for (i = 0; i < n_vertices; i++)
    {
        dp_matrix[i] = (int *)malloc(n_vertices * sizeof(int));
    }

    generate_random_adj_matrix(n_vertices);

    /* Call parallel implementations */
    floyd_warshall_parallel(adjacency_matrix, dp_matrix, n_vertices);

    /* Free allocated memory */
    for (int i = 0; i < n_vertices; i++)
    {
        free(dp_matrix[i]);
    }

    free(dp_matrix);

    dp_matrix = (int **)malloc(n_vertices * sizeof(int *));
    for (i = 0; i < n_vertices; i++)
    {
        dp_matrix[i] = (int *)malloc(n_vertices * sizeof(int));
    }

    /* Call serial implementation */
    floyd_warshall_serial(adjacency_matrix, dp_matrix, n_vertices);

    /* Free allocated memory */
    for (int i = 0; i < n_vertices; i++)
    {
        free(dp_matrix[i]);
        free(adjacency_matrix[i]);
    }

    free(dp_matrix);
    free(adjacency_matrix);

    return 0;
}
